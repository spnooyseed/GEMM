#include "stdio.h"
#include <hip/hip_runtime.h>

#include <time.h>
#define N 32 * 1024 * 1024
#define BLOCK_SIZE 256

__global__ void reduce_v1_no_divergence(float *g_idata, float *g_odata) {
  __shared__ float sdata[BLOCK_SIZE];

  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i];
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    int index = 2 * s * tid;
    if (index < blockDim.x) {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0)
    g_odata[blockIdx.x] = sdata[0];
}

bool check(float *output_host) {
  for (int i = 0; i < N / BLOCK_SIZE; ++i) {
    if (output_host[i] != BLOCK_SIZE * 2.0) {
      return false;
    }
  }
  return true;
}
int main() {
  float *input_host = (float *)malloc(N * sizeof(float));
  float *input_device;
  hipMalloc((void **)&input_device, N * sizeof(float));
  for (int i = 0; i < N; i++)
    input_host[i] = 2.0;
  hipMemcpy(input_device, input_host, N * sizeof(float),
             hipMemcpyHostToDevice);

  int32_t block_num = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  float *output_host = (float *)malloc((N / BLOCK_SIZE) * sizeof(float));
  float *output_device;
  hipMalloc((void **)&output_device, (N / BLOCK_SIZE) * sizeof(float));

  dim3 grid(N / BLOCK_SIZE, 1);
  dim3 block(BLOCK_SIZE, 1);
  hipEvent_t start, stop;
  float msecTotal = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  int nIter = 1000;
  for (int i = 1; i <= nIter; ++i) {
    reduce_v1_no_divergence<<<grid, block>>>(input_device, output_device);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&msecTotal, start, stop);
  hipMemcpy(output_host, output_device, block_num * sizeof(float),
             hipMemcpyDeviceToHost);
  msecTotal /= nIter;
  if (check(output_host)) {
    // bandwidth maybe error
    printf("reduce_v1_no_divergence , use time = %.3fms , and bandwidth = "
           "%.3fGB/s\n",
           msecTotal, N * sizeof(float) * 1e-9 / (msecTotal * 1e-3f));
  } else {
    printf("reduce_v1_no_divergence , this is error\n");
  }

  return 0;
}