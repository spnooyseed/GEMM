#include "stdio.h"
#include "stdlib.h"

#include "assert.h"

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <iostream>
#define checkCudaErrors(func)                                                  \
  {                                                                            \
    hipError_t e = func;                                                      \
    if (e != hipSuccess) {                                                    \
      printf("gemm_v2_shared_block , %s , %d CUDA: %s\n", __FILE__, __LINE__,  \
             hipGetErrorString(e));                                           \
    }                                                                          \
  }

// transfer float4
#define Fetch_Float4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
#define Offset(row, col, k) ((row) * (k) + (col))
template <
    const int Block_Size_N, /*height of block of C that each thread block
                               calculate*/
    const int Block_Size_K, /*width of block of A that each thread block load
                         into shared memory*/

    const int Block_Size_M,  /*width of block of C that each thread block
                                calculate*/
    const int Thread_Size_Y, /*height of block of C that each thread calculate*/
    const int Thread_Size_X, /* width of block of C that each thread calculate*/
    const bool double_buffer /*whether enable double buffering or not*/>

__global__ void Gemm(float *__restrict__ A, float *__restrict__ B,
                     float *__restrict__ C, const int N, const int M,
                     const int K) {
  // block index
  int bx = blockIdx.x, by = blockIdx.y;
  // thread index
  int tx = threadIdx.x, ty = threadIdx.y;

  __shared__ float As[Block_Size_K][Block_Size_N];
  __shared__ float Bs[Block_Size_K][Block_Size_M];
  const int loadN = 4;

  A = &A[Offset(by * Block_Size_N, 0, K)];
  B = &B[Offset(0, bx * Block_Size_M, M)];

  // const int thread_num_per_block =
  //     Block_Size_N / Thread_Size_Y * (Block_Size_M / Thread_Size_X);
  // const int tid = ty * blockDim.x + tx;
  const int thread_num_x_per_block = Block_Size_N / Thread_Size_Y,
            thread_num_y_per_block = Block_Size_M / Thread_Size_X;
  const int thread_num_per_block =
      thread_num_x_per_block * thread_num_y_per_block;
  const int tid = ty * thread_num_x_per_block + tx;

  float accum[Thread_Size_Y][Thread_Size_X] = {0};

  float reg_A[Thread_Size_Y], reg_B[Thread_Size_X];

  const int load_num_A = Block_Size_N * Block_Size_K / thread_num_per_block;

  float load_A_reg[load_num_A];

  const int A_Tile_Thread_Per_Row = Block_Size_K / loadN,
            B_Tile_Thread_Per_Row = Block_Size_M / loadN;

  const int A_Tile_Row = tid / A_Tile_Thread_Per_Row,
            B_Tile_Row = tid / B_Tile_Thread_Per_Row;
  const int A_Tile_Col = tid % A_Tile_Thread_Per_Row * loadN,
            B_Tile_Col = tid % B_Tile_Thread_Per_Row * loadN;
  const int A_Tile_Row_Stride = thread_num_per_block / A_Tile_Thread_Per_Row,
            B_Tile_Row_Stride = thread_num_per_block / B_Tile_Thread_Per_Row;

  for (int tile_idx_k = 0; tile_idx_k < K; tile_idx_k += Block_Size_K) {
    // store transpose shared_memory matrix As from global memory
    // load A from global to shared
    for (int tile_idx_bn = 0; tile_idx_bn < Block_Size_N;
         tile_idx_bn += A_Tile_Row_Stride) {
      int index = tile_idx_bn / A_Tile_Row_Stride * loadN;
      Fetch_Float4(load_A_reg[index]) = Fetch_Float4(
          A[Offset(A_Tile_Row + tile_idx_bn, A_Tile_Col + tile_idx_k, K)]);
      for (int load_N = 0; load_N < loadN; ++load_N) {
        As[A_Tile_Col + load_N][A_Tile_Row + tile_idx_bn] =
            load_A_reg[index + load_N];
      }
    }
    // load B from global to shared
    for (int tile_idx_bk = 0; tile_idx_bk < Block_Size_K;
         tile_idx_bk += B_Tile_Row_Stride) {
      Fetch_Float4(Bs[B_Tile_Row + tile_idx_bk][B_Tile_Col]) = Fetch_Float4(
          B[Offset(tile_idx_k + tile_idx_bk + B_Tile_Row, B_Tile_Col, M)]);
    }
    __syncthreads();

    for (int tile_idx_bk = 0; tile_idx_bk < Block_Size_K; ++tile_idx_bk) {
      // load A from shared to register
      for (int tile_idx_rn = 0; tile_idx_rn < Thread_Size_Y;
           tile_idx_rn += loadN) {
        Fetch_Float4(reg_A[tile_idx_rn]) =
            Fetch_Float4(As[tile_idx_bk][Thread_Size_Y * ty + tile_idx_rn]);
      }
      // load B from shared to register
      for (int tile_idx_rm = 0; tile_idx_rm < Thread_Size_X;
           tile_idx_rm += loadN) {
        Fetch_Float4(reg_B[tile_idx_rm]) =
            Fetch_Float4(Bs[tile_idx_bk][Thread_Size_X * tx + tile_idx_rm]);
      }
      // compute accum(RN , RM) by (reg_A , reg_B)
      for (int tile_idx_rn = 0; tile_idx_rn < Thread_Size_Y; ++tile_idx_rn) {
        for (int tile_idx_rm = 0; tile_idx_rm < Thread_Size_X; ++tile_idx_rm) {
          accum[tile_idx_rn][tile_idx_rm] +=
              reg_A[tile_idx_rn] * reg_B[tile_idx_rm];
        }
      }
    }
    __syncthreads();
  }
  // store back to C
  for (int tile_idx_rn = 0; tile_idx_rn < Thread_Size_Y; ++tile_idx_rn) {
    for (int tile_idx_rm = 0; tile_idx_rm < Thread_Size_X;
         tile_idx_rm += loadN) {
      Fetch_Float4(
          C[Offset(by * Block_Size_N + Thread_Size_Y * ty + tile_idx_rn,
                   bx * Block_Size_M + Thread_Size_X * tx + tile_idx_rm, M)]) =
          Fetch_Float4(accum[tile_idx_rn][tile_idx_rm]);
    }
  }
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("gemm_v2_shared_block , usage: ./main [N] [K] [M]");
    exit(0);
  }
  size_t N = atoi(argv[1]), K = atoi(argv[2]), M = atoi(argv[3]);

  float *h_a = new float[N * K], *h_b = new float[K * M],
        *h_c = new float[N * M], *h_c1 = new float[N * M];

  float *d_a, *d_b, *d_c;
  size_t bytes_A = sizeof(float) * N * K, bytes_B = sizeof(float) * M * K,
         bytes_C = sizeof(float) * N * M;
  checkCudaErrors(hipMalloc(&d_a, sizeof(float) * N * K));
  checkCudaErrors(hipMalloc(&d_b, sizeof(float) * M * K));
  checkCudaErrors(hipMalloc(&d_c, sizeof(float) * N * M));

  const int Block_Size_N = 128, Block_Size_M = 128, Block_Size_K = 8,
            Thread_Size_X = 8, Thread_Size_Y = 8;
  const bool Enable_Double_Buffer = false;
  // generate A
  for (int i = 0; i < N * K; ++i) {
    h_a[i] = i;
  }
  // generate B
  for (int i = 0; i < M * K; ++i) {
    h_b[i] = i;
  }
  checkCudaErrors(hipMemcpy(d_a, h_a, bytes_A, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_b, h_b, bytes_B, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float msecTotal = 0;
  int nIter = 1000;

  checkCudaErrors(hipMemcpy(d_c, h_c, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));

  for (int run = 0; run < nIter; ++run) {
    dim3 dimBlock(Block_Size_N / Thread_Size_Y, Block_Size_M / Thread_Size_X);
    dim3 dimGrid(N / Block_Size_N, M / Block_Size_M);
    Gemm<Block_Size_N, Block_Size_K, Block_Size_M, Thread_Size_Y, Thread_Size_X,
         Enable_Double_Buffer><<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N, M, K);
  }

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  checkCudaErrors(hipMemcpy(h_c, d_c, bytes_C, hipMemcpyDeviceToHost));
  // puts("generate A");
  // // generate A
  // for (int i = 0; i < N * K; ++i) {
  //   if (i % K == 0)
  //     puts("");
  //   std::cout << h_a[i] << " ";
  // }
  // puts("\ngenerate B");
  // // generate B
  // for (int i = 0; i < M * K; ++i) {
  //   if (i % M == 0)
  //     puts("");
  //   std::cout << h_b[i] << " ";
  // }

  // puts("\nNative Gemm output = \n");
  // for (int i = 0; i < N; ++i) {
  //   for (int j = 0; j < M; ++j) {
  //     double ans = 0;
  //     for (int k = 0; k < K; ++k) {
  //       ans += h_a[i * K + k] * h_b[k * M + j];
  //     }
  //     printf("%.0f ", ans);
  //   }
  //   puts("");
  // }
  // puts("\n MyGemm output = ");
  // for (int j = 0; j < N; ++j) {
  //   for (int k = 0; k < M; ++k) {
  //     printf("%.0f ", h_c[j * M + k]);
  //   }
  //   printf("\n");
  // }

  double msecPerMatrixMul[2] = {0, 0};
  double gigaFlops[2] = {0, 0};
  double flopsPerMatrixMul = 2.0 * M * N * K;
  msecPerMatrixMul[0] = msecTotal / nIter;
  gigaFlops[0] =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
  printf("gemm_v2_shared_block , My gemm Performance= %.2f GFlop/s, Time= %.3f "
         "msec, Size= %.0f Ops,\n",
         gigaFlops[0], msecPerMatrixMul[0], flopsPerMatrixMul);

  // cublas
  hipblasHandle_t blas_handle;
  hipblasCreate(&blas_handle);
  float alpha = 1.0;
  float beta = 0;
  checkCudaErrors(hipMemcpy(d_c, h_c, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));
  for (int run = 0; run < nIter; run++) {
    hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K, &alpha, d_a, K,
                d_b, M, &beta, d_c, N);
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  checkCudaErrors(hipMemcpy(h_c1, d_c, bytes_C, hipMemcpyDeviceToHost));
  // puts("cublas");
  // for (int j = 0; j < N; ++j) {
  //   for (int k = 0; k < M; ++k) {
  //     //   // printf("%d %d" , j , k) ;
  //     printf("%.0f ", h_c1[j * M + k]);
  //     // C[j][k] = As[k][j] ;
  //   }
  //   printf("\n");
  // }
  msecPerMatrixMul[1] = msecTotal / nIter;
  gigaFlops[1] =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
  printf("gemm_v2_shared_block , CuBlas Performance= %.2f GFlop/s, Time= %.3f "
         "msec, Size= %.0f Ops,\n",
         gigaFlops[1], msecPerMatrixMul[1], flopsPerMatrixMul);

  hipblasDestroy(blas_handle);

  double eps = 1.e-6; // machine zero
  bool correct = true;
  for (int i = 0; i < N * M; i++) {
    int row = i / M;
    int col = i % M;
    double abs_err = fabs(h_c[i] - h_c1[col * N + row]);
    double dot_length = M;
    double abs_val = fabs(h_c[i]);
    double rel_err = abs_err / abs_val / dot_length;
    if (rel_err > eps) {
      printf("gemm_v2_shared_block , Error! Matrix[%05d]=%.8f, ref=%.8f error "
             "term is > %E\n",
             i, h_c[i], h_c1[col * N + row], eps);
      correct = false;
      break;
    }
  }

  printf("gemm_v2_shared_block , %s\n",
         correct ? "Result= PASS" : "Result= FAIL");
  printf("gemm_v2_shared_block , ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

  delete h_a, h_b, h_c, h_c1;
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}