#include "stdio.h"
#include "stdlib.h"

#include "assert.h"

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <iostream>
#define checkCudaErrors(func)                                                  \
  {                                                                            \
    hipError_t e = func;                                                      \
    if (e != hipSuccess) {                                                    \
      printf("gemm_v2_shared_block , %s , %d CUDA: %s\n", __FILE__, __LINE__,  \
             hipGetErrorString(e));                                           \
    }                                                                          \
  }

// transfer float4
#define Fetch_Float4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
#define Offset(row, col, k) ((row) * (k) + (col))
template <
    const int Block_Size_N, /*height of block of C that each thread block
                               calculate*/
    const int Block_Size_K, /*width of block of A that each thread block load
                         into shared memory*/

    const int Block_Size_M,  /*width of block of C that each thread block
                                calculate*/
    const int Thread_Size_X, /*height of block of C that each thread calculate*/
    const int Thread_Size_Y, /* width of block of C that each thread calculate*/
    const bool double_buffer /*whether enable double buffering or not*/>

__global__ void Gemm(float *__restrict__ A, float *__restrict__ B,
                     float *__restrict__ C, const int N, const int M,
                     const int K) {
  // block index
  int bx = blockIdx.x, by = blockIdx.y;
  // thread index
  int tx = threadIdx.x, ty = threadIdx.y;

  int row = by * blockDim.y + ty, col = bx * blockDim.x + tx;
  __shared__ float As[Block_Size_N][Block_Size_K];
  __shared__ float Bs[Block_Size_K][Block_Size_M];
  float tmp = 0.0;
  for (int i = 0; i < K; i += Block_Size_K) {
    // store transpose shared_memory matrix As from global memory
    As[tx][ty] = A[Offset(row, tx + i, K)];
    Bs[ty][tx] = B[Offset(ty + i, col, M)];
    __syncthreads();
    for (int j = 0; j < Block_Size_K; ++j) {
      // load transpose shared_memory matrix As
      tmp += As[j][ty] * Bs[j][tx];
    }
  }
  C[Offset(row, col, M)] = tmp;
}
int main(int argc, char **argv) {
  if (argc != 4) {
    printf("gemm_v2_shared_block , usage: ./main [N] [K] [M]");
    exit(0);
  }
  size_t N = atoi(argv[1]), K = atoi(argv[2]), M = atoi(argv[3]);

  float *h_a = new float[N * K], *h_b = new float[K * M],
        *h_c = new float[N * M], *h_c1 = new float[N * M];

  float *d_a, *d_b, *d_c;
  size_t bytes_A = sizeof(float) * N * K, bytes_B = sizeof(float) * M * K,
         bytes_C = sizeof(float) * N * M;
  checkCudaErrors(hipMalloc(&d_a, sizeof(float) * N * K));
  checkCudaErrors(hipMalloc(&d_b, sizeof(float) * M * K));
  checkCudaErrors(hipMalloc(&d_c, sizeof(float) * N * M));

  const int Block_Size_N = 32, Block_Size_M = 32, Block_Size_K = 32,
            Thread_Size_X = 8, Thread_Size_Y = 8;
  const bool Enable_Double_Buffer = false;
  // generate A
  for (int i = 0; i < N * K; ++i) {
    h_a[i] = i;
  }
  // generate B
  for (int i = 0; i < M * K; ++i) {
    h_b[i] = i;
  }
  checkCudaErrors(hipMemcpy(d_a, h_a, bytes_A, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_b, h_b, bytes_B, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float msecTotal = 0;
  int nIter = 1000;

  checkCudaErrors(hipMemcpy(d_c, h_c, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));

  for (int run = 0; run < nIter; ++run) {
    dim3 dimBlock(Block_Size_N, Block_Size_M);
    // size_t block_num_x= N / Block_Size_N , block_num_y = M / Block_Size_M ;
    dim3 dimGrid(N / Block_Size_N, M / Block_Size_M);
    // printf("Grid = (%d , %d)\n" , dimGrid.x , dimGrid.y) ;
    Gemm<Block_Size_N, Block_Size_K, Block_Size_M, Thread_Size_X, Thread_Size_Y,
         Enable_Double_Buffer><<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N, M, K);
  }

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  checkCudaErrors(hipMemcpy(h_c, d_c, bytes_C, hipMemcpyDeviceToHost));
  // puts("generate A");
  // // generate A
  // for (int i = 0; i < N * K; ++i) {
  //   if (i % K == 0)
  //     puts("");
  //   std::cout << h_a[i] << " ";
  // }
  // puts("\ngenerate B");
  // // generate B
  // for (int i = 0; i < M * K; ++i) {
  //   if (i % M == 0)
  //     puts("");
  //   std::cout << h_b[i] << " ";
  // }

  // puts("\nNative Gemm output = \n");
  // for (int i = 0; i < N; ++i) {
  //   for (int j = 0; j < M; ++j) {
  //     double ans = 0;
  //     for (int k = 0; k < K; ++k) {
  //       ans += h_a[i * K + k] * h_b[k * M + j];
  //     }
  //     printf("%.0f ", ans);
  //   }
  //   puts("");
  // }
  // puts("\n MyGemm output = ");
  // for (int j = 0; j < N; ++j) {
  //   for (int k = 0; k < M; ++k) {
  //     printf("%.0f ", h_c[j * M + k]);
  //   }
  //   printf("\n");
  // }

  double msecPerMatrixMul[2] = {0, 0};
  double gigaFlops[2] = {0, 0};
  double flopsPerMatrixMul = 2.0 * M * N * K;
  msecPerMatrixMul[0] = msecTotal / nIter;
  gigaFlops[0] =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
  printf("gemm_v2_shared_block , My gemm Performance= %.2f GFlop/s, Time= %.3f "
         "msec, Size= %.0f Ops,\n",
         gigaFlops[0], msecPerMatrixMul[0], flopsPerMatrixMul);

  // cublas
  hipblasHandle_t blas_handle;
  hipblasCreate(&blas_handle);
  float alpha = 1.0;
  float beta = 0;
  checkCudaErrors(hipMemcpy(d_c, h_c, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));
  for (int run = 0; run < nIter; run++) {
    hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K, &alpha, d_a, K,
                d_b, M, &beta, d_c, N);
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  checkCudaErrors(hipMemcpy(h_c1, d_c, bytes_C, hipMemcpyDeviceToHost));
  // puts("cublas");
  // for (int j = 0; j < N; ++j) {
  //   for (int k = 0; k < M; ++k) {
  //     //   // printf("%d %d" , j , k) ;
  //     printf("%.0f ", h_c1[j * M + k]);
  //     // C[j][k] = As[k][j] ;
  //   }
  //   printf("\n");
  // }
  msecPerMatrixMul[1] = msecTotal / nIter;
  gigaFlops[1] =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
  printf("gemm_v2_shared_block , CuBlas Performance= %.2f GFlop/s, Time= %.3f "
         "msec, Size= %.0f Ops,\n",
         gigaFlops[1], msecPerMatrixMul[1], flopsPerMatrixMul);

  hipblasDestroy(blas_handle);

  double eps = 1.e-6; // machine zero
  bool correct = true;
  for (int i = 0; i < N * M; i++) {
    int row = i / M;
    int col = i % M;
    double abs_err = fabs(h_c[i] - h_c1[col * N + row]);
    double dot_length = M;
    double abs_val = fabs(h_c[i]);
    double rel_err = abs_err / abs_val / dot_length;
    if (rel_err > eps) {
      printf("gemm_v2_shared_block , Error! Matrix[%05d]=%.8f, ref=%.8f error "
             "term is > %E\n",
             i, h_c[i], h_c1[col * N + row], eps);
      correct = false;
      break;
    }
  }

  printf("gemm_v2_shared_block , %s\n",
         correct ? "Result= PASS" : "Result= FAIL");
  printf("gemm_v2_shared_block , ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

  delete h_a, h_b, h_c, h_c1;
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}