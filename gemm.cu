#include "stdio.h"
#include "stdlib.h"

#include "assert.h"

#include "hipblas.h"
#include "hip/hip_runtime.h"
#include <iostream>
#define checkCudaErrors(func)                                                  \
  {                                                                            \
    hipError_t e = func;                                                      \
    if (e != hipSuccess) {                                                    \
      printf("%s , %d CUDA: %s\n", __FILE__, __LINE__, hipGetErrorString(e)); \
    }                                                                          \
  }

// transfer float4
#define Fetch_Float4(pointer) (reinterpret_cast<float4 *>(&(pointer))[0])
#define Offset(row, col, k) ((row) * (k) + (col))
template <
    const int Block_Size_N, /*height of block of C that each thread block
                               calculate*/
    const int Block_Size_K, /*width of block of A that each thread block load
                         into shared memory*/

    const int Block_Size_M,  /*width of block of C that each thread block
                                calculate*/
    const int Thread_Size_X, /*height of block of C that each thread calculate*/
    const int Thread_Size_Y, /* width of block of C that each thread calculate*/
    const bool double_buffer /*whether enable double buffering or not*/>

__global__ void Gemm(float *__restrict__ A, float *__restrict__ B,
                     float *__restrict__ C, const int N, const int M,
                     const int K) {
  // block index
  int bx = blockIdx.x, by = blockIdx.y;
  // thread index
  int tx = threadIdx.x, ty = threadIdx.y;

  // X、Y threads number in block
  const int thread_num_x_per_block = Block_Size_N / Thread_Size_X;
  const int thread_num_y_per_block = Block_Size_M / Thread_Size_Y;
  const int thread_num_per_block =
      thread_num_x_per_block * thread_num_y_per_block;

  // thread Id in cur Block
  // const int tid = ty * thread_num_y_per_block + tx;
  const int tid = tx * thread_num_y_per_block + ty;

  // shared memory , A_shared , B_shared
  __shared__ float As[Block_Size_K][Block_Size_N];
  __shared__ float Bs[Block_Size_K][Block_Size_M];
  // register for C
  float accum[Thread_Size_X][Thread_Size_Y] = {0};
  // register for A , B
  float reg_a[Thread_Size_X], reg_b[Thread_Size_Y];

  const int load_num = 4;
  // resgister for load global memory
  const int load_num_A =
      Block_Size_N * Block_Size_K / (thread_num_per_block * load_num);
  const int load_num_B =
      Block_Size_K * Block_Size_M / (thread_num_per_block * load_num);

  float load_A_reg[load_num * load_num_A], load_B_reg[load_num * load_num_B];
  // threads num in one row
  const int A_Tile_Thread_Per_Row = Block_Size_K / load_num,
            B_Tile_Thread_Per_Row = Block_Size_M / load_num;

  const int A_Tile_Row = tid / A_Tile_Thread_Per_Row,
            B_Tile_Row = tid / B_Tile_Thread_Per_Row;
  const int A_Tile_Col = tid % A_Tile_Thread_Per_Row * load_num;
  const int B_Tile_Col = tid % B_Tile_Thread_Per_Row * load_num;
  const int A_Tile_Row_Stride = thread_num_per_block / A_Tile_Thread_Per_Row;
  const int B_Tile_Row_Stride = thread_num_per_block / B_Tile_Thread_Per_Row;

  A = &A[Block_Size_N * bx * K];
  B = &B[Block_Size_M * by];
#pragma unroll
  for (int i = 0; i < K; i += Block_Size_K) {
#pragma unroll
    // load A、B from global to shared memory
    for (int j = 0; j < Block_Size_N; j += A_Tile_Row_Stride) {
      int load_index = j / A_Tile_Row_Stride * load_num;
#pragma unroll
      for (int loadN = 0; loadN < load_num; ++loadN) {
        load_A_reg[load_index + loadN] =
            A[Offset(A_Tile_Row + j, A_Tile_Col + i, K) + loadN];
      }
#pragma unroll
      for (int loadN = 0; loadN < load_num; ++loadN) {
        As[A_Tile_Col + loadN][A_Tile_Row + j] = load_A_reg[load_index + loadN];
      }
    }
#pragma unroll
    for (int j = 0; j < Block_Size_K; j += B_Tile_Row_Stride) {
#pragma unroll
      for (int loadN = 0; loadN < load_num; ++loadN) {
        Bs[B_Tile_Row + j][B_Tile_Col + loadN] =
            B[Offset(B_Tile_Row + i + j, B_Tile_Col, M) + loadN];
      }
    }
    __syncthreads();
// load A、B from shared_memory to register
#pragma unroll
    for (int j = 0; j < Block_Size_K; ++j) {
#pragma unroll
      for (int thread_x = 0; thread_x < Thread_Size_X; thread_x += load_num) {
#pragma unroll
        for (int loadN = 0; loadN < load_num; ++loadN) {
          reg_a[thread_x + loadN] =
              As[j][Thread_Size_X * tx + thread_x + loadN];
        }
      }
#pragma unroll
      for (int thread_y = 0; thread_y < Thread_Size_Y; thread_y += load_num) {
#pragma unroll
        for (int loadN = 0; loadN < load_num; ++loadN) {
          reg_b[thread_y + loadN] =
              Bs[j][Thread_Size_Y * ty + thread_y + loadN];
        }
      }
#pragma unroll
      for (int thread_x = 0; thread_x < Thread_Size_X; ++thread_x) {
#pragma unroll
        for (int thread_y = 0; thread_y < Thread_Size_Y; ++thread_y) {
          accum[thread_x][thread_y] += reg_a[thread_x] * reg_b[thread_y];
        }
      }
    }
#pragma unroll
    for (int thread_x = 0; thread_x < Thread_Size_X; thread_x++) {
#pragma unroll
      for (int thread_y = 0; thread_y < Thread_Size_Y; thread_y += load_num) {
#pragma unroll
        for (int loadN = 0; loadN < load_num; ++loadN) {

          C[Offset(Block_Size_N * bx + tx * Thread_Size_X + thread_x,
                   Block_Size_M * by + ty * Thread_Size_Y + thread_y, M) +
            loadN] = accum[thread_x][thread_y + loadN];
        }
      }
    }
  }
}

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("usage: ./main [N] [K] [M]");
    exit(0);
  }
  size_t N = atoi(argv[1]), K = atoi(argv[2]), M = atoi(argv[3]);

  float *h_a = new float[N * K], *h_b = new float[K * M],
        *h_c = new float[N * M], *h_c1 = new float[N * M];

  float *d_a, *d_b, *d_c;
  size_t bytes_A = sizeof(float) * N * K, bytes_B = sizeof(float) * M * K,
         bytes_C = sizeof(float) * N * M;
  checkCudaErrors(hipMalloc(&d_a, sizeof(float) * N * K));
  checkCudaErrors(hipMalloc(&d_b, sizeof(float) * M * K));
  checkCudaErrors(hipMalloc(&d_c, sizeof(float) * N * M));

  const int Block_Size_N = 128, Block_Size_M = 128, Block_Size_K = 8,
            Thread_Size_X = 8, Thread_Size_Y = 8;
  const bool Enable_Double_Buffer = false;
  // generate A
  for (int i = 0; i < N * K; ++i) {
    h_a[i] = i;
  }
  // generate B
  for (int i = 0; i < M * K; ++i) {
    h_b[i] = i;
  }
  checkCudaErrors(hipMemcpy(d_a, h_a, bytes_A, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_b, h_b, bytes_B, hipMemcpyHostToDevice));

  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float msecTotal = 0;
  int nIter = 1000;

  checkCudaErrors(hipMemcpy(d_c, h_c, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));

  for (int run = 0; run < nIter; ++run) {
    dim3 dimBlock(Block_Size_N / Thread_Size_X, Block_Size_M / Thread_Size_Y);
    dim3 dimGrid(N / Block_Size_N, M / Block_Size_M);
    Gemm<Block_Size_N, Block_Size_K, Block_Size_M, Thread_Size_X, Thread_Size_Y,
         Enable_Double_Buffer><<<dimGrid, dimBlock>>>(d_a, d_b, d_c, N, M, K);
  }

  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));
  checkCudaErrors(hipMemcpy(h_c, d_c, bytes_C, hipMemcpyDeviceToHost));
  // puts("generate A");
  // // generate A
  // for (int i = 0; i < N * K; ++i) {
  //   if (i % K == 0)
  //     puts("");
  //   std::cout << h_a[i] << " ";
  // }
  // puts("\ngenerate B");
  // // generate B
  // for (int i = 0; i < M * K; ++i) {
  //   if (i % M == 0)
  //     puts("");
  //   std::cout << h_b[i] << " ";
  // }

  // puts("\nNative Gemm output = \n");
  // for (int i = 0; i < N; ++i) {
  //   for (int j = 0; j < M; ++j) {
  //     double ans = 0;
  //     for (int k = 0; k < K; ++k) {
  //       ans += h_a[i * K + k] * h_b[k * M + j];
  //     }
  //     printf("%.0f ", ans);
  //   }
  //   puts("");
  // }
  // puts("\n MyGemm output = ");
  // for (int j = 0; j < N; ++j) {
  //   for (int k = 0; k < M; ++k) {
  //     printf("%.0f ", h_c[j * M + k]);
  //   }
  //   printf("\n");
  // }

  double msecPerMatrixMul[2] = {0, 0};
  double gigaFlops[2] = {0, 0};
  double flopsPerMatrixMul = 2.0 * M * N * K;
  msecPerMatrixMul[0] = msecTotal / nIter;
  gigaFlops[0] =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
  printf(
      "My gemm Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
      gigaFlops[0], msecPerMatrixMul[0], flopsPerMatrixMul);

  // cublas
  hipblasHandle_t blas_handle;
  hipblasCreate(&blas_handle);
  float alpha = 1.0;
  float beta = 0;
  checkCudaErrors(hipMemcpy(d_c, h_c, bytes_C, hipMemcpyHostToDevice));
  checkCudaErrors(hipEventRecord(start));
  for (int run = 0; run < nIter; run++) {
    hipblasSgemm(blas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, M, K, &alpha, d_a, K,
                d_b, M, &beta, d_c, N);
  }
  checkCudaErrors(hipEventRecord(stop));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  checkCudaErrors(hipMemcpy(h_c1, d_c, bytes_C, hipMemcpyDeviceToHost));
  // puts("cublas");
  // for (int j = 0; j < N; ++j) {
  //   for (int k = 0; k < M; ++k) {
  //     //   // printf("%d %d" , j , k) ;
  //     printf("%.0f ", h_c1[j * M + k]);
  //     // C[j][k] = As[k][j] ;
  //   }
  //   printf("\n");
  // }
  msecPerMatrixMul[1] = msecTotal / nIter;
  gigaFlops[1] =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[1] / 1000.0f);
  printf("CuBlas Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,\n",
         gigaFlops[1], msecPerMatrixMul[1], flopsPerMatrixMul);

  hipblasDestroy(blas_handle);

  double eps = 1.e-6; // machine zero
  bool correct = true;
  for (int i = 0; i < N * M; i++) {
    int row = i / M;
    int col = i % M;
    double abs_err = fabs(h_c[i] - h_c1[col * N + row]);
    double dot_length = M;
    double abs_val = fabs(h_c[i]);
    double rel_err = abs_err / abs_val / dot_length;
    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i,
             h_c[i], h_c1[col * N + row], eps);
      correct = false;
      break;
    }
  }

  printf("%s\n", correct ? "Result= PASS" : "Result= FAIL");
  printf("ratio= %f\n", gigaFlops[0] / gigaFlops[1]);

  delete h_a, h_b, h_c, h_c1;
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}